
#include <hip/hip_runtime.h>
#include <cinttypes>
#include <stdio.h>

#define MAT_SIZE_X 10000
#define MAT_SIZE_Y 10000

int main(void) {
    uint32_t mat_size_x = MAT_SIZE_X;
    uint32_t mat_size_y = MAT_SIZE_Y;
    int nBytes = sizeof(float) * mat_size_x * mat_size_y;

    float *hMat_A;
    float *hMat_B;
    float *hMat_G;

    hMat_A = (float *) malloc(nBytes);
    hMat_B = (float *) malloc(nBytes);
    hMat_G = (float *) malloc(nBytes);

    // TODO

    free(hMat_A);
    free(hMat_B);
    free(hMat_G);
}

#include <cinttypes>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <>

#define LOOP 10

#define BLOCK_SIZE 32

#define MAT_SIZE_X 10000
#define MAT_SIZE_Y 10000

#define CHECK(func)                   \
{                                     \
  const hipError_t error = func;     \
  if (error != hipSuccess) {         \
    printf("Error: %s:%d\n", __FILE__, __LINE__); \
    printf("Code: %d, Reason: %s\n", error, hipGetErrorString(error)); \
    hipDeviceReset();                \
    exit(EXIT_FAILURE);               \
  }                                   \
}

void calculate_gpu(float *hMat_A, float *hMat_B, float *hMat_G, uint32_t mat_size_x, uint32_t mat_size_y) {
    float *dMat_A = NULL;
    float *dMat_B = NULL;
    float *dMat_G = NULL;
    int nBytes = sizeof(float) * mat_size_x * mat_size_y;

    CHECK(hipMalloc((float **) &dMat_A, nBytes));
    CHECK(hipMalloc((float **) &dMat_B, nBytes));
    CHECK(hipMalloc((float **) &dMat_G, nBytes));

    CHECK(hipMemcpy(dMat_A, hMat_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dMat_B, hMat_B, nBytes, hipMemcpyHostToDevice));

    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((mat_size_x + block.x - 1) / block.x, (mat_size_y + block.y - 1) / block.y);
    printf("Grid=(%d, %d), Block=(%d,%d)\n", grid.x, grid.y, block.x, block.y);
	
    // TODO

    CHECK(hipMemcpy(hMat_G, dMat_G, nBytes, hipMemcpyDeviceToHost));

    CHECK(hipFree(dMat_A));
    CHECK(hipFree(dMat_B));
    CHECK(hipFree(dMat_G));
    CHECK(hipDeviceReset());
}

void add_vector_cpu(float *hMat_A, float *hMat_B, float *hMat_G, uint32_t mat_size_x, uint32_t mat_size_y) {
    for (uint32_t y = 0; y < mat_size_y; y++) {
       for (uint32_t x = 0; x < mat_size_x; x++) {
           uint32_t index = y * mat_size_x + x;
           hMat_G[index] = hMat_A[index] + hMat_B[index];
       }
    }
}

// CPUで A+B=G を実装
void calculate_cpu(float *hMat_A, float *hMat_B, float *hMat_G, uint32_t mat_size_x, uint32_t mat_size_y) {
    for (int i = 0; i < LOOP; i++) {
        add_vector_cpu(hMat_A, hMat_B, hMat_G, mat_size_x, mat_size_y);
    }
}

int main(void) {
    uint32_t mat_size_x = MAT_SIZE_X;
    uint32_t mat_size_y = MAT_SIZE_Y;
    int nBytes = sizeof(float) * mat_size_x * mat_size_y;

    float *hMat_A;
    float *hMat_B;
    float *hMat_G;

    hMat_A = (float *) malloc(nBytes);
    hMat_B = (float *) malloc(nBytes);
    hMat_G = (float *) malloc(nBytes);

    // 乱数で行列Aと行列Bを初期化する
    time_t t;
    srand((unsigned int) time(&t));
    for (uint32_t i = 0; i < mat_size_x * mat_size_y; i++) {
        hMat_A[i] = (float)(rand() % 100000) / 10000.0f;
	hMat_B[i] = (float)(rand() % 100000) / 10000.0f;
    }

    // calculate_cpu(hMat_A, hMat_B, hMat_G, mat_size_x, mat_size_y);
    calculate_gpu(hMat_A, hMat_B, hMat_G, mat_size_x, mat_size_y);

    free(hMat_A);
    free(hMat_B);
    free(hMat_G);
}
